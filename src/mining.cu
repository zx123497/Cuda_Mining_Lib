#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

#define ROTLEFT(a, b) ((a << b) | (a >> (32 - b)))
#define ROTRIGHT(a, b) ((a >> b) | (a << (32 - b)))

#define CH(x, y, z) ((x & y) ^ (~x & z))
#define MAJ(x, y, z) ((x & y) ^ (x & z) ^ (y & z))
#define EP0(x) (ROTRIGHT(x, 2) ^ ROTRIGHT(x, 13) ^ ROTRIGHT(x, 22))
#define EP1(x) (ROTRIGHT(x, 6) ^ ROTRIGHT(x, 11) ^ ROTRIGHT(x, 25))
#define SIG0(x) (ROTRIGHT(x, 7) ^ ROTRIGHT(x, 18) ^ (x >> 3))
#define SIG1(x) (ROTRIGHT(x, 17) ^ ROTRIGHT(x, 19) ^ (x >> 10))

#define BLOCK_SIZE 256

__constant__ uint32_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

__device__ void sha256_transform(uint32_t *state, const unsigned char data[])
{
    uint32_t a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
    for (; i < 64; ++i)
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

    a = state[0];
    b = state[1];
    c = state[2];
    d = state[3];
    e = state[4];
    f = state[5];
    g = state[6];
    h = state[7];

    for (i = 0; i < 64; ++i)
    {
        t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
        t2 = EP0(a) + MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

    state[0] += a;
    state[1] += b;
    state[2] += c;
    state[3] += d;
    state[4] += e;
    state[5] += f;
    state[6] += g;
    state[7] += h;
}

__device__ void sha256_init(uint32_t *state)
{
    state[0] = 0x6a09e667;
    state[1] = 0xbb67ae85;
    state[2] = 0x3c6ef372;
    state[3] = 0xa54ff53a;
    state[4] = 0x510e527f;
    state[5] = 0x9b05688c;
    state[6] = 0x1f83d9ab;
    state[7] = 0x5be0cd19;
}

__device__ void sha256_update(uint32_t *state, const unsigned char *data, size_t len, unsigned char *buffer, size_t *bitlen)
{
    size_t i;

    for (i = 0; i < len; ++i)
    {
        buffer[*bitlen >> 3] = data[i];
        *bitlen += 8;
        if (*bitlen == 512)
        {
            sha256_transform(state, buffer);
            *bitlen = 0;
        }
    }
}

__device__ void sha256_final(uint32_t *state, unsigned char *buffer, size_t *bitlen, unsigned char *hash)
{
    size_t i;

    i = *bitlen >> 3;
    buffer[i++] = 0x80;
    while (i < 56)
        buffer[i++] = 0x00;

    for (int j = 56; j < 64; ++j)
    {
        buffer[j] = 0;
    }

    sha256_transform(state, buffer);

    for (i = 0; i < 4; ++i)
    {
        hash[i] = (state[0] >> (24 - i * 8)) & 0xff;
        hash[i + 4] = (state[1] >> (24 - i * 8)) & 0xff;
        hash[i + 8] = (state[2] >> (24 - i * 8)) & 0xff;
        hash[i + 12] = (state[3] >> (24 - i * 8)) & 0xff;
        hash[i + 16] = (state[4] >> (24 - i * 8)) & 0xff;
        hash[i + 20] = (state[5] >> (24 - i * 8)) & 0xff;
        hash[i + 24] = (state[6] >> (24 - i * 8)) & 0xff;
        hash[i + 28] = (state[7] >> (24 - i * 8)) & 0xff;
    }
}

// CUDA kernel to compute the hash
__global__ void mine_kernel(const char *block_data, int block_data_len, int difficulty, unsigned char *valid_hash, int *nonce_found, int max_nonce)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= max_nonce)
        return;

    // Initialize SHA-256 state
    uint32_t state[8];
    sha256_init(state);

    // Copy the block data and append the nonce
    unsigned char buffer[64] = {0};
    size_t bitlen = 0;
    sha256_update(state, (unsigned char *)block_data, block_data_len, buffer, &bitlen);
    sha256_update(state, (unsigned char *)&idx, sizeof(idx), buffer, &bitlen);

    // Finalize the hash computation
    unsigned char hash[32];
    sha256_final(state, buffer, &bitlen, hash);

    // Check if the hash is valid by looking for leading zeros
    bool valid = true;
    for (int i = 0; i < difficulty; ++i)
    {
        if (hash[i] != 0)
        {
            valid = false;
            break;
        }
    }

    // If the hash is valid, store the result and the nonce
    if (valid && atomicCAS(nonce_found, -1, idx) == -1)
    {
        memcpy(valid_hash, hash, 32);
    }
}

// Host function to launch the kernel and handle the mining
extern "C" void mine_block(const char *block_data, int block_data_len, int difficulty, unsigned char *valid_hash, int *nonce_found, int max_nonce)
{
    char *d_block_data;
    unsigned char *d_valid_hash;
    int *d_nonce_found;

    hipMalloc(&d_block_data, block_data_len);
    hipMalloc(&d_valid_hash, 32 * sizeof(unsigned char));
    hipMalloc(&d_nonce_found, sizeof(int));

    hipMemcpy(d_block_data, block_data, block_data_len, hipMemcpyHostToDevice);
    hipMemset(d_nonce_found, -1, sizeof(int));

    int num_blocks = (max_nonce + BLOCK_SIZE - 1) / BLOCK_SIZE;
    mine_kernel<<<num_blocks, BLOCK_SIZE>>>(d_block_data, block_data_len, difficulty, d_valid_hash, d_nonce_found, max_nonce);

    hipMemcpy(valid_hash, d_valid_hash, 32 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(nonce_found, d_nonce_found, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_block_data);
    hipFree(d_valid_hash);
    hipFree(d_nonce_found);
}
